#include "hip/hip_runtime.h"

#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>
#include <string.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;
    std::string inputFilename2;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);


        //For Blending

        if (argc == 4){
     


        	inputFilename2 = std::string(argv[3]);
        	labwork.loadInputImage2(inputFilename2);


                //Resizing

                int w = labwork.getWidth();
                int h = labwork.getHeight();

                
		char * cmd = (char *)malloc(sizeof(char)*100); ;
                sprintf(cmd, "convert %s -resize %dx%d %s",argv[3], w, h, argv[3]);
		const char * command = cmd;




                system(command);

                labwork.loadInputImage2(inputFilename2);
 
        }
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
	    printf("labwork 1 OPENMP ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:

            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
	    printf("labwork 3 ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            printf("labwork 4 GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            break;
        case 5:
            labwork.labwork5_GPU();
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::loadInputImage2(std::string inputFileName) {
    inputImage2 = jpegLoader.load(inputFileName);
}

int Labwork::getWidth() {
	return inputImage->width;
}

int Labwork::getHeight() {
	return inputImage->height;
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));

	#pragma omp parallel for

        for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    int nDevices = 0;
    // get all devices
    hipGetDeviceCount(&nDevices);
    printf("Number total of GPU : %d\n\n", nDevices);
    for (int i = 0; i < nDevices; i++){
        // get informations from individual device
        hipDeviceProp_t prop;
	printf("Filling GPU number : %d\n", i);
        hipGetDeviceProperties(&prop, i);
        // something more here		

		//Core INFO
		printf("Device name of GPU number %d : %s\n", i, prop.name);
		printf("Clock rate: %d\n",prop.clockRate);
		int nbCores = getSPcores(prop);
		printf("Number of cores: %d\n", nbCores);
		printf("Number of multiprocessors on device : %d\n", prop.multiProcessorCount);
		printf("Warp Size : %d\n", prop.warpSize);

		//Memory INFO
		printf("Memory Clock Rate : %d\n",prop.memoryClockRate);
		printf("Memory Bus Width : %d\n", prop.memoryBusWidth);
		
		
	
    }

}

//Write a grey scale kernel here :
 __global__ void grayScale(uchar3 *input, uchar3 *output) {
       int tid = threadIdx.x + blockIdx.x * blockDim.x;
       output[tid].x = (input[tid].x + input[tid].y +
                       input[tid].z) / 3;
       output[tid].z = output[tid].y = output[tid].x;
}
//This should be executed on a device core.

void Labwork::labwork3_GPU() {
    // Calculate number of pixels
 
    int pixelCount = inputImage->width * inputImage->height ;

    outputImage = static_cast<char *>(malloc(pixelCount * 3));

    //Allocate CUDA Memory

    uchar3 *devInput;
    uchar3 *devOutput;
    hipMalloc(&devInput, pixelCount *sizeof(uchar3));
    hipMalloc(&devOutput, pixelCount *sizeof(uchar3));


    //Copy CUDA Memory from CPU to GPU

    hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);

    //Processing

    int blockSize = 64;
    int numBlock = pixelCount  / blockSize;
    printf("numblock %d\n", numBlock);
    grayScale<<<numBlock, blockSize>>>(devInput , devOutput);
    
    //Copy CUDA Memory from GPU to CPU

    hipMemcpy(outputImage, devOutput, pixelCount * sizeof(uchar3), hipMemcpyDeviceToHost);

    //Cleaning

    hipFree(devInput);
    hipFree(devOutput);


}


//Write a grey scale kernel here :
 __global__ void grayScale2(uchar3 *input, uchar3 *output,int width, int height) {

       int x = threadIdx.x + blockIdx.x * blockDim.x;
       int y = threadIdx.y + blockIdx.y * blockDim.y;
       int w = blockDim.x * gridDim.x;

       //if ((gridDim.x * gridDim.y) < width * height){
       
       	int tid = y*w + x; 

        output[tid].x = (input[tid].x + input[tid].y +
                       input[tid].z) / 3;
        output[tid].z = output[tid].y = output[tid].x;

      // }
}

void Labwork::labwork4_GPU() {

    // Calculate number of pixels
 
    int pixelCount = inputImage->width * inputImage->height ;

    outputImage = static_cast<char *>(malloc(pixelCount * 3));

    //Allocate CUDA Memory

    uchar3 *devInput;
    uchar3 *devOutput;
    hipMalloc(&devInput, pixelCount *sizeof(uchar3));
    hipMalloc(&devOutput, pixelCount *sizeof(uchar3));


    //Copy CUDA Memory from CPU to GPU

    hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);

    //Processing

    dim3 blockSize = dim3(8, 8);
//    int rx = inputImage->width%blockSize.x;
//    int ry = inputImage->height%blockSize.y;
    dim3 gridSize = dim3 (inputImage->width/blockSize.x,inputImage->height/blockSize.y);  
    grayScale2<<<gridSize, blockSize>>>(devInput, devOutput, inputImage->width, inputImage->height);    

    //Copy CUDA Memory from GPU to CPU

    hipMemcpy(outputImage, devOutput, pixelCount * sizeof(uchar3), hipMemcpyDeviceToHost);

    //Cleaning

    hipFree(devInput);
    hipFree(devOutput);


}

//Write a grey scale kernel here :
 __global__ void grayScale3(uchar3 *input, uchar3 *output,int width, int height) {

        
       int x = threadIdx.x + blockIdx.x * blockDim.x;
       int y = threadIdx.y + blockIdx.y * blockDim.y;

       //if ((gridDim.x * gridDim.y) < width * height){
       
        int tid = y*width + x; 

        if (x<width){

        if (y<height){ 


        output[tid].x = (input[tid].x + input[tid].y +
                       input[tid].z) / 3;

        output[tid].z = output[tid].y = output[tid].x;
        }
        }

      // }
}

//Write a grey scale kernel here :
 __global__ void blur(uchar3 *input, uchar3 *output,int width, int height) {

       int matrix[7][7] = {{0,0,1,2,1,0,0},{0,3,13,22,13,3,0},{1,3,59,97,59,13,1},{2,22,97,159,97,22,2},{1,3,59,97,59,3,1},{0,3,13,22,13,3,0},{0,0,1,2,1,0,0}};


       int x = threadIdx.x + blockIdx.x * blockDim.x;
       int y = threadIdx.y + blockIdx.y * blockDim.y;

       //if ((gridDim.x * gridDim.y) < width * height){
   
        int tid = y*width + x; 

        int outputTemp = 0;

        int sommeCoef = 0;

        if (x<width){ 

        if (y<height){  

        if (x>3 && x<width-3 && y>3 && y<height-3){ 

                for (int i=0; i<7; i++){

                	for (int j=0; j<7; j++){

        			outputTemp += input[(y-3+i)*width+(x-3+j)].x*matrix[j][i]; 
                                
                                sommeCoef += matrix[j][i];

                        }

                        
                        output[tid].x = outputTemp / sommeCoef;

   			output[tid].z = output[tid].y = output[tid].x;

		}
        }

        }

        }

      // }
}






  void Labwork::labwork5_CPU() {

  // Calculate number of pixels
 
    int pixelCount = inputImage->width * inputImage->height ;

    outputImage = static_cast<char *>(malloc(pixelCount * 3));

    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (( (int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }

       int matrix[7][7] = {{0,0,1,2,1,0,0},{0,3,13,22,13,3,0},{1,3,59,97,59,13,1},{2,22,97,159,97,22,2},{1,3,59,97,59,3,1},{0,3,13,22,13,3,0},{0,0,1,2,1,0,0}};



       int outputTemp = 0;

       int sommeCoef = 0;

       int width = inputImage->width; 

       for (int k = 0; k< inputImage->height;k++){ 

       for (int l = 0; l< width; l++){

                for (int i=0; i<7; i++){

                        for (int j=0; j<7; j++){

                                outputTemp += ((int) inputImage->buffer[(k-3+i)*width+(l-3+j)] ) *matrix[j][i]; 
                                
                                sommeCoef += matrix[j][i];

                        }

                        
                        outputImage[i*3] = (char) (outputTemp / sommeCoef);
                        outputImage[i * 3 + 1] = outputImage[i * 3];
                        outputImage[i * 3 + 2] = outputImage[i * 3];

                }

        }

        }




} 


//Labwork5_GPU() without shared memory.

void Labwork::labwork5_GPU() {



  // Calculate number of pixels
 

    int pixelCount = inputImage->width * inputImage->height ;


    outputImage = static_cast<char *>(malloc(pixelCount * 3));

    //Allocate CUDA Memory

    uchar3 *devInput;
    uchar3 *devOutput;
    uchar3 *devGray;
    hipMalloc(&devInput, pixelCount *sizeof(uchar3));
    hipMalloc(&devOutput, pixelCount *sizeof(uchar3));
    hipMalloc(&devGray, pixelCount *sizeof(uchar3));

    //Copy CUDA Memory from CPU to GPU

    hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);

    //Processing

    dim3 blockSize = dim3(32, 32);

//    int rx = inputImage->width%blockSize.x;
//    int ry = inputImage->height%blockSize.y;

    int numBlockx = inputImage-> width / (blockSize.x) ;
    int numBlocky = inputImage-> height / (blockSize.y) ;
    if ((inputImage-> width % (blockSize.x)) > 0) {
    	numBlockx++ ;                                                                
    }                 
    if ((inputImage-> height % (blockSize.y)) > 0){ 
        numBlocky++ ;                                            
    }                                                             

    dim3 gridSize = dim3 (numBlockx,numBlocky);  
    grayScale3<<<gridSize, blockSize>>>(devInput, devGray, inputImage->width, inputImage->height);    
    blur<<<gridSize, blockSize>>>(devGray, devOutput, inputImage->width, inputImage->height);

    //Copy CUDA Memory from GPU to CPU

    hipMemcpy(outputImage, devOutput, pixelCount * sizeof(uchar3), hipMemcpyDeviceToHost);

    //Cleaning

    hipFree(devInput);
    hipFree(devOutput);

}

 __global__ void binary(uchar3 *input, uchar3 *output,int width, int height,int threshold) {

        
       int x = threadIdx.x + blockIdx.x * blockDim.x;
       int y = threadIdx.y + blockIdx.y * blockDim.y;

       
        int tid = y*width + x; 

        if (x<width){

        if (y<height){ 

        output[tid].x = (input[tid].x + input[tid].y +
                       input[tid].z) / 3;


        if (output[tid].x >= threshold){

                output[tid].x = 255;
        	output[tid].z = output[tid].y = output[tid].x;

        }

        else {
	        output[tid].x = 0;
        	output[tid].z = output[tid].y = output[tid].x;
        }

        }
        }

      // }
}

 __global__ void brightness(uchar3 *input, uchar3 *output,int width, int height,int brightness) {

        
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;
       
        int tid = y*width + x; 

        if (x<width){

        if (y<height){ 

        output[tid].x = (input[tid].x + input[tid].y +
                       input[tid].z) / 3;

        if (brightness> 50 && brightness != 100){

        	if (output[tid].x + brightness <= 255){ 

        		output[tid].x += brightness;

        	}

        	else{
        
        		output[tid].x = 255;

        	}

        }

        if (brightness < 50 && brightness !=0){
	
	  if (output[tid].x - brightness >= 0){ 

                        output[tid].x -= (50-brightness);

                }

                else{
        
                        output[tid].x = 0;

                }



        }

        if (brightness == 100){
    
            output[tid].x = 255;
        }

        if (brightness == 0){ 
    
            output[tid].x = 0;
        }

        output[tid].z = output[tid].y = output[tid].x;

        }
        }

}

 __global__ void blendingGray(uchar3 *input, uchar3 *input2, uchar3 *output,int width, int height,float coefficient) {

        
       int x = threadIdx.x + blockIdx.x * blockDim.x;
       int y = threadIdx.y + blockIdx.y * blockDim.y;

       
        int tid = y*width + x; 

        int nbPixels = width * height;
	float prod = coefficient * (float) nbPixels;
        int prodfin = (int) prod;


        if (x<width){

        if (y<height){ 

        if (tid <= prodfin){

        	output[tid].x = input[tid].x;

        	output[tid].z = output[tid].y = output[tid].x;

        }
        else{

        	output[tid].x = input2[tid].x; 

                output[tid].z = output[tid].y = output[tid].x;

        }

        }
        }

      
}


void Labwork::labwork6_GPU() {

      // Calculate number of pixels
 

    int pixelCount = inputImage->width * inputImage->height ;


    outputImage = static_cast<char *>(malloc(pixelCount * 3));

    //Allocate CUDA Memory

    uchar3 *devInput;
    uchar3 *devOutput;
    uchar3 *devGray;
    hipMalloc(&devInput, pixelCount *sizeof(uchar3));
    hipMalloc(&devOutput, pixelCount *sizeof(uchar3));
    hipMalloc(&devGray, pixelCount *sizeof(uchar3));

    //For Blending
    uchar3 *devInput2;
    uchar3 *devGray2;
    hipMalloc(&devInput2, pixelCount *sizeof(uchar3));
    hipMalloc(&devGray2, pixelCount *sizeof(uchar3));

    //Copy CUDA Memory from CPU to GPU

    hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);

    //hipMemcpy(devInput2, inputImage2->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);

    //Processing

    dim3 blockSize = dim3(32, 32);

    int threshold = 128;

    int brightnessVar = 50; //choose your brightness between 0 and 100. A value of 50 will leave the brightness unchanged.
    
    float coefficient = 0.5;  //Blending Coefficient.

    int numBlockx = inputImage-> width / (blockSize.x) ;
    int numBlocky = inputImage-> height / (blockSize.y) ;
    if ((inputImage-> width % (blockSize.x)) > 0) {
        numBlockx++ ;
    }
    if ((inputImage-> height % (blockSize.y)) > 0){
        numBlocky++ ;
    }

    dim3 gridSize = dim3 (numBlockx,numBlocky);  

    grayScale3<<<gridSize, blockSize>>>(devInput, devGray, inputImage->width, inputImage->height);

    hipFree(devInput);

    hipMemcpy(devInput2, inputImage2->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);

    grayScale3<<<gridSize, blockSize>>>(devInput2, devGray2, inputImage->width, inputImage->height);

    blendingGray<<<gridSize, blockSize>>>(devGray, devGray2, devOutput, inputImage->width, inputImage->height, coefficient);

    //Copy CUDA Memory from GPU to CPU

    hipMemcpy(outputImage, devOutput, pixelCount * sizeof(uchar3), hipMemcpyDeviceToHost);

    //Cleaning

    //hipFree(devInput);
    hipFree(devInput2);
    hipFree(devOutput);

}

void Labwork::labwork7_GPU() {
}

void Labwork::labwork8_GPU() {
}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU(){
}


























